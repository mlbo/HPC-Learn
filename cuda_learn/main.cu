
#include <hip/hip_runtime.h>
#include <stdio.h>

void cpu() {
    printf("Hello cpu\n");
}

__global__ void gpu() {
    printf("Hello gpu\n");
}

int main() {
    cpu();
    gpu<<< 1, 1>>>();
    hipDeviceSynchronize();

    return 0;

}